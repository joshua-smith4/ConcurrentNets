/*
 * GPUKernels.cu
 *
 *  Created on: Oct 19, 2010
 *      Author: yiding
 */

#include <hip/hip_runtime.h>
#include <math.h>


#define NOID   0xFFFFFFFF

#define CUDA_MAJOR_VER 1
#define CUDA_MINOR_VER 3


typedef unsigned int CoordType;
typedef unsigned int IdType;
typedef unsigned short CapType;
typedef unsigned int SizeType;
typedef float CostType;

// template <unsigned N>
// struct ParallelHistAccumulator
// {
// public:
//   unsigned* bins[N];
//   unsigned numBins;
//   inline __device__ void init(unsigned numberOfBins)
//   {
//     numThreadBins = num;
//     for(auto i = 0u; i < N; ++i)
//     {
//       bins[i] = new unsigned[numBins];
//     }
//   }
//   inline __device__
//   inline __device__ void cleanup()
//   {
//     for(auto i = 0u; i < N; ++i)
//     {
//       delete[] bins[i];
//     }
//   }
//   inline __device__ unsigned incrementBin(unsigned bin)
//   {
//     return 0u;
//   }
//   inline __device__ unsigned* getHist()
//   {
//
//   }
// };

//Define CUDA Kernels in this file
__global__ void colorTiles_noshared(IdType** colorTiles, uint2* a, uint2* b, unsigned subNetCount, int yTiles, int xTiles, int minY, int maxY, int minX, int maxX)
{

}

__global__ void countSubnets_noshared(IdType** colorTiles, unsigned* tilesWithinRoutingRegion, int minY, int maxY, int minX, int maxX)
{

}
