/*
 * GPUKernels.cu
 *
 *  Created on: Oct 19, 2010
 *      Author: yiding
 */

#include <hip/hip_runtime.h>
#include <math.h>


#define NOID   0xFFFFFFFF

#define CUDA_MAJOR_VER 1
#define CUDA_MINOR_VER 3


typedef unsigned int CoordType;
typedef unsigned int IdType;
typedef unsigned short CapType;
typedef unsigned int SizeType;
typedef float CostType;

//Define CUDA Kernels in this file
__global__ void prep_noshared()
{
  
}
