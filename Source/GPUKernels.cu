/*
 * GPUKernels.cu
 *
 *  Created on: Oct 19, 2010
 *      Author: yiding
 */

#include <hip/hip_runtime.h>
#include <math.h>


#define NOID   0xFFFFFFFF

#define CUDA_MAJOR_VER 1
#define CUDA_MINOR_VER 3


typedef unsigned int CoordType;
typedef unsigned int IdType;
typedef unsigned short CapType;
typedef unsigned int SizeType;
typedef float CostType;

//Define CUDA Kernels in this file
__global__ void colorTiles_noshared(unsigned* colorTiles, size_t pitchColorTiles, uint2* a, uint2* b, unsigned subNetCount, int minY, int maxY, int minX, int maxX)
{
  int x = blockIdx.x * blockDim.x + threadIdx.x + minX;
  int y = blockIdx.y * blockDim.y + threadIdx.y + minY;
  if (y >= minY && y <= maxY && x >= minX && x <= maxX)
  {
    IdType* elem = (IdType*)((char*)colorTiles + y * pitchColorTiles) + x;
    for(int i = 0; i < subNetCount; ++i)
    {
      if (a[i].x <= x && b[i].x >= x && a[i].y <= y && b[i].y >= y)
      {
        *elem = i;
        break;
      }
    }
  }
}


__global__ void histCalc_noshared(unsigned* tilesWithinRoutingRegion, size_t pitchTiles, IdType* colorTiles, size_t pitchColor, unsigned subNetCount, int minY, int maxY, int minX, int maxX, unsigned num_concurrency_bins)
{
  int x = blockIdx.x * blockDim.x + threadIdx.x + minX;
  int y = blockIdx.y * blockDim.y + threadIdx.y + minY;
  if (y >= minY && y <= maxY && x >= minX && x <= maxX)
  {
    IdType* elem = (IdType*)((char*)colorTiles + y * pitchColor) + x;
    if(*elem != NOID)
    {
      unsigned* tileElem = (unsigned*)((char*)tilesWithinRoutingRegion + ((blockDim.x*gridDim.x*(blockIdx.y*blockDim.y+threadIdx.y)+threadIdx.x) % num_concurrency_bins) * pitchTiles) + (*elem);
      atomicAdd(tileElem, 1);
    }
  }
}

__global__ void sumHist_noshared(unsigned* tilesWithinRoutingRegion, size_t pitchTiles, unsigned* returnVal, unsigned subNetCount, unsigned num_concurrency_bins)
{
  int i = threadIdx.x;
  if(i < subNetCount)
  {
    for(int j = 0; j < num_concurrency_bins; ++j)
    {
      unsigned* elem = (unsigned*)((char*)tilesWithinRoutingRegion + j * pitchTiles) + i;
      returnVal[i] += *elem;
    }
  }
}
